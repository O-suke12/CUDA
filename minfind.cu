
#include <hip/hip_runtime.h>
#include <stdio.h>
#define ROWS 8
#define COLUMNS 1000000

__global__ void minfind(int*a, int*c)
{
	int min = 1000000001;
	int x = threadIdx.x;
	for(int i=0; i<COLUMNS; i++){
		if(a[x*COLUMNS+i]<min){
			min=a[x*COLUMNS+i];
		}
	}
	c[x]=min;
}

int main(){
	int* dev_a,* dev_c, * a=(int*)malloc(ROWS*COLUMNS*sizeof(int));
	int  c[ROWS], seq_min=1000000000, min=1000000000;
	
	hipMalloc((void**)&dev_a, ROWS*COLUMNS*sizeof(int));
	hipMalloc((void**)&dev_c, 8*sizeof(int));
	
	for (int y=0; y<ROWS; y++){
		for (int x=0; x<COLUMNS; x++){
			a[y*COLUMNS+x] = rand()%1000000001;
		}
		hipMemcpy(dev_a, a, ROWS*COLUMNS*sizeof(int), hipMemcpyHostToDevice);
	}
	minfind <<<1, ROWS>>>(dev_a,dev_c);
	hipMemcpy(c, dev_c, 8*sizeof(int), hipMemcpyDeviceToHost);
	
	for (int i=0; i<ROWS; i++)
		for(int j=0; j<COLUMNS; j++)
			if(a[i*COLUMNS+j]<seq_min)
				seq_min=a[i*COLUMNS+j];
 
	for (int i=0; i<ROWS; i++)
		if(c[i]<min)
			min=c[i];
	printf("Sequential search: %d, Thread search: %d\n",seq_min,min);
	
	hipFree(dev_a); hipFree(dev_c);
	return 0;
}

